#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <float.h>
#include "neutral.h"
#include "neutral.k"
#include "../neutral_interface.h"
#include "../../comms.h"
#include "../../shared.h"
#include "../../shared_data.h"
#include "../../params.h"

#if 0
#ifdef MPI
#include "mpi.h"
#endif
#endif // if 0

// Performs a solve of dependent variables for particles transport.
void solve_transport_2d(
    const int nx, const int ny, const int global_nx, const int global_ny, 
    const int x_off, const int y_off, const double dt, const int nparticles_total,
    int* nlocal_particles, uint64_t* master_key, const int* neighbours, 
    Particles* particles, const double* density, const double* edgex, 
    const double* edgey, const double* edgedx, const double* edgedy, 
    CrossSection* cs_scatter_table, CrossSection* cs_absorb_table, 
    double* scalar_flux_tally, double* energy_deposition_tally, RNPool* rn_pools,
    int* reduce_array0, int* reduce_array1)
{
  // This is the known starting number of particles
  uint64_t facets = 0;
  uint64_t collisions = 0;
  int nparticles = *nlocal_particles;
  int nparticles_sent[NNEIGHBOURS];

  if(!nparticles) {
    printf("out of particles\n");
    return;
  }

  // Communication isn't required for edges
  for(int ii = 0; ii < NNEIGHBOURS; ++ii) {
    nparticles_sent[ii] = 0;
  }

  handle_particles(
      global_nx, global_ny, nx, ny, x_off, y_off, dt, neighbours, density, edgex, 
      edgey, &facets, &collisions, nparticles_sent, master_key, nparticles_total, 
      nparticles, &nparticles, particles, cs_scatter_table, 
      cs_absorb_table, scalar_flux_tally, energy_deposition_tally, rn_pools,
      reduce_array0, reduce_array1);

  *nlocal_particles = nparticles;

  printf("facets %llu collisions %llu\n", facets, collisions);
}

// Handles the current active batch of particles
void handle_particles(
    const int global_nx, const int global_ny, const int nx, const int ny, 
    const int x_off, const int y_off, const double dt, const int* neighbours, 
    const double* density, const double* edgex, const double* edgey, uint64_t* facets, 
    uint64_t* collisions, int* nparticles_sent, uint64_t* master_key, 
    const int nparticles_total, const int nparticles_to_process, 
    int* nparticles, Particles* particles, CrossSection* cs_scatter_table, 
    CrossSection* cs_absorb_table, double* scalar_flux_tally, 
    double* energy_deposition_tally, RNPool* rn_pools, int* reduce_array0,
    int* reduce_array1)
{
  int nthreads                  = 0;

  int nparticles_out = 0;
  int nparticles_dead = 0;

  hipDeviceSynchronize();

  // Block over the events
  const int block_size = nparticles_total;
  const int nblocks = ceil(nparticles_total/(double)block_size);
  for(int bb = 0; bb < nblocks; ++bb) {
    const int particles_offset = bb*block_size;

    int initialised = 0;

    while(1) {
      START_PROFILING(&compute_profile);
      update_rn_pool_master_keys(rn_pools, 1, (*master_key)++);
      STOP_PROFILING(&compute_profile, "update rn pool master keys");

      /* INITIALISATION */
      if(!initialised) {
        START_PROFILING(&compute_profile);
        event_initialisation(
            block_size, particles_offset, nx, x_off, y_off, particles, dt, 
            density, nthreads, rn_pools, cs_scatter_table, cs_absorb_table);
        initialised = 1;
        STOP_PROFILING(&compute_profile, "initialisation");
      }

      START_PROFILING(&compute_profile);
      const int all_census = calc_next_event(
          block_size, particles_offset, particles, facets, collisions, 
          reduce_array0, reduce_array1, x_off, y_off, edgex, edgey);
      STOP_PROFILING(&compute_profile, "calc next event");

      if(all_census) {
        break;
      }

      START_PROFILING(&compute_profile);
      handle_facets(
          block_size, particles_offset, global_nx, global_ny, nx, ny, x_off, 
          y_off, neighbours, nparticles_sent, particles, edgex, edgey, density, 
          &nparticles_out, scalar_flux_tally, energy_deposition_tally,
          cs_scatter_table, cs_absorb_table);
      STOP_PROFILING(&compute_profile, "handle facets");

      START_PROFILING(&compute_profile);
      handle_collisions( 
          block_size, particles_offset, nx, x_off, y_off, particles, edgex, edgey, 
          rn_pools, &nparticles_dead, cs_scatter_table, cs_absorb_table,
          scalar_flux_tally, energy_deposition_tally, reduce_array0);
      STOP_PROFILING(&compute_profile, "handle collisions");
    }

    START_PROFILING(&compute_profile);
    handle_census(
        block_size, particles_offset, nx, x_off, y_off, particles, density, edgex, 
        edgey, cs_scatter_table, cs_absorb_table, scalar_flux_tally, 
        energy_deposition_tally);
    STOP_PROFILING(&compute_profile, "handle census");
  }

  printf("handled %d particles, with %d particles deleted\n", 
      nparticles_to_process, nparticles_dead+nparticles_out);
}

// Initialises ready for the event cycles
void event_initialisation(
    const int nparticles, const int particles_offset, const int nx, 
    const int x_off, const int y_off, Particles* particles, const double dt, 
    const double* density, const int nthreads, RNPool* rn_pools, 
    CrossSection* cs_scatter_table, CrossSection* cs_absorb_table)
{
  RNPool* master_pool = &rn_pools[0];

  // Initialise all of the particles with their starting state
#if 0
  const int nblocks = ceil(nparticles/((double)NTHREADS*NRANDOM_NUMBERS)); 
#endif // if 0
  const int nblocks = ceil(nparticles/((double)NTHREADS)); 
  event_initialisation_kernel<<<nblocks, NTHREADS>>>(
      nparticles, particles_offset, dt, nx, x_off, y_off, cs_scatter_table->nentries, 
      cs_absorb_table->nentries, particles->e, particles->cellx, 
      particles->celly, cs_scatter_table->keys, cs_scatter_table->values, 
      cs_absorb_table->keys, cs_absorb_table->values, 
      cs_scatter_table->log_width, cs_absorb_table->log_width,
      density, particles->dt_to_census, 
      particles->next_event, particles->scatter_cs_index, 
      particles->absorb_cs_index, particles->particle_velocity, 
      particles->local_density, particles->cell_mfp, particles->mfp_to_collision,
      master_pool->key.v[0], particles->energy_deposition);  

  // TODO: BE CAREFUL PASSING MASTER KEY HERE, MAKE SURE IT IS INITIALISED
  // PROPERLY ETC..
}

// Calculates the next event for each particle
int calc_next_event(
    const int nparticles, const int particles_offset, Particles* particles, 
    uint64_t* facets, uint64_t* collisions, int* reduce_array0, int* reduce_array1,
    const int x_off, const int y_off, const double* edgex, const double* edgey)
{
  /* CALCULATE THE EVENTS */
  const int nblocks = ceil(nparticles/(double)NTHREADS); 
  calc_next_event_kernel<<<nblocks, NTHREADS>>>(
      nparticles, particles_offset, particles->mfp_to_collision, 
      particles->cell_mfp, particles->particle_velocity, particles->dt_to_census, 
      particles->distance_to_facet, particles->next_event, reduce_array0,
      reduce_array1, particles->e, x_off, y_off, particles->x, particles->y, 
      particles->omega_x, particles->omega_y, particles->x_facet, 
      particles->cellx, particles->celly, particles->scatter_cs_index, 
      particles->absorb_cs_index, edgex, edgey);

  int nfacets = 0;
  int ncollisions = 0;
  finish_sum_int_reduce(nblocks, reduce_array0, &ncollisions);
  finish_sum_int_reduce(nblocks, reduce_array1, &nfacets);
  *facets += nfacets;
  *collisions += ncollisions;

#if 0
  printf("calculated the events collision %d facets %d census/dead %d\n",
      ncollisions, nfacets, (nparticles-nfacets-ncollisions));
#endif // if 0

  return (!nfacets && !ncollisions);
}

// Handle all of the facet encounters
void handle_facets(
    const int nparticles, const int particles_offset, const int global_nx, 
    const int global_ny, const int nx, const int ny, const int x_off, 
    const int y_off, const int* neighbours, int* nparticles_sent, 
    Particles* particles, const double* edgex, const double* edgey, 
    const double* density, int* nparticles_out, double* scalar_flux_tally, 
    double* energy_deposition_tally, CrossSection* cs_scatter_table, 
    CrossSection* cs_absorb_table)
{
  const int nthreads = NTHREADS;
  const int nblocks = ceil(nparticles/(double)NTHREADS); 
  handle_facets_kernel<<<nblocks, nthreads>>>(
      nparticles, particles_offset, global_nx, global_ny, nx, x_off, y_off, 
      cs_scatter_table->nentries, cs_absorb_table->nentries, particles->e, 
      particles->distance_to_facet, particles->weight, cs_scatter_table->keys, 
      cs_scatter_table->values, cs_absorb_table->keys, cs_absorb_table->values, 
      cs_scatter_table->log_width, cs_absorb_table->log_width, density, 
      particles->energy_deposition, particles->x, particles->y, particles->omega_x, 
      particles->omega_y, particles->x_facet, particles->cellx, particles->celly,
      particles->dt_to_census, particles->next_event, particles->scatter_cs_index,
      particles->absorb_cs_index, particles->particle_velocity, particles->local_density,
      particles->cell_mfp, particles->mfp_to_collision, energy_deposition_tally);

  *nparticles_out = 0;
}

// Handle all of the collision events
void handle_collisions(
    const int nparticles, const int particles_offset, const int nx, 
    const int x_off, const int y_off, Particles* particles, const double* edgex, 
    const double* edgey, RNPool* rn_pools, int* nparticles_dead, 
    CrossSection* cs_scatter_table, CrossSection* cs_absorb_table, 
    double* scalar_flux_tally, double* energy_deposition_tally, int* reduce_array)
{
  RNPool* master_pool = &rn_pools[0];

  int np_dead = 0;

  const int nblocks = ceil(nparticles/(double)NTHREADS); 
  handle_collisions_kernel<<<nblocks, NTHREADS>>>(
      nparticles, particles_offset, nx, x_off, y_off, 
      cs_scatter_table->nentries, cs_absorb_table->nentries, particles->e, 
      particles->distance_to_facet, particles->weight, cs_scatter_table->keys, 
      cs_scatter_table->values, cs_absorb_table->keys, cs_absorb_table->values,
      cs_scatter_table->log_width, cs_absorb_table->log_width,
      particles->energy_deposition, particles->x, particles->y, 
      particles->omega_x, particles->omega_y, particles->x_facet, particles->cellx, 
      particles->celly, particles->dt_to_census, particles->next_event, 
      particles->scatter_cs_index, particles->absorb_cs_index, 
      particles->particle_velocity, particles->local_density, particles->cell_mfp, 
      particles->mfp_to_collision, reduce_array, master_pool->key.v[0],
      energy_deposition_tally);

  finish_sum_int_reduce(nblocks, reduce_array, &np_dead);
  *nparticles_dead += np_dead;
}

// Handles all of the census events
void handle_census(
    const int nparticles, const int particles_offset, const int nx, 
    const int x_off, const int y_off, Particles* particles, const double* density, 
    const double* edgex, const double* edgey, CrossSection* cs_scatter_table, 
    CrossSection* cs_absorb_table, double* scalar_flux_tally, 
    double* energy_deposition_tally)
{
  /* HANDLE THE CENSUS EVENTS */
  const int nblocks = ceil(nparticles/(double)NTHREADS); 
  handle_census_kernel<<<nblocks, NTHREADS>>>(
      nparticles, nx, x_off, y_off, particles_offset, particles->next_event, 
      particles->particle_velocity, particles->dt_to_census, particles->cellx, 
      particles->celly, particles->local_density, particles->e, 
      particles->scatter_cs_index, particles->absorb_cs_index, particles->x, 
      particles->y, particles->omega_x, particles->omega_y, particles->mfp_to_collision, 
      particles->energy_deposition, density, cs_scatter_table->keys, 
      cs_absorb_table->keys, cs_scatter_table->values, cs_absorb_table->values,
      cs_scatter_table->log_width, cs_absorb_table->log_width,
      cs_scatter_table->nentries, cs_absorb_table->nentries, particles->weight,
      energy_deposition_tally);
}

// Sends a particles to a neighbour and replaces in the particles list
void send_and_mark_particle(
    const int destination, const int pindex, Particles* particles)
{
#ifdef MPI
  if(destination == EDGE) {
    return;
  }

  particles->next_event[pindex] = 1;

#if 0
  // Send the particles
  MPI_Send(
      particles, 1, particle_type, destination, TAG_PARTICLE, MPI_COMM_WORLD);
#endif // if 0
#else
  TERMINATE("Unreachable - shouldn't send particles unless MPI enabled.\n");
#endif
}

// Acts as a particle source
void inject_particles(
    Mesh* mesh, const int local_nx, const int local_ny, 
    const double local_particle_left_off, const double local_particle_bottom_off,
    const double local_particle_width, const double local_particle_height, 
    const int nparticles, const double initial_energy, RNPool* rn_pools,
    Particles* particles)
{
  RNPool* master_pool = &rn_pools[0];

  START_PROFILING(&compute_profile);

  const int nblocks = ceil(nparticles/(double)NTHREADS); 
  inject_particles_kernel<<<nblocks, NTHREADS>>>(
      local_nx, local_ny, mesh->x_off, mesh->y_off, local_particle_left_off, 
      local_particle_bottom_off, local_particle_width, local_particle_height, 
      nparticles, mesh->dt, initial_energy, master_pool->key.v[0], mesh->edgex, mesh->edgey, 
      particles->x, particles->y, particles->cellx, particles->celly, 
      particles->omega_x, particles->omega_y, particles->e, particles->weight, 
      particles->dt_to_census, particles->mfp_to_collision, 
      particles->scatter_cs_index, particles->absorb_cs_index, particles->next_event);

  STOP_PROFILING(&compute_profile, "initialising particles");
}

// Validates the results of the simulation
void validate(
    const int nx, const int ny, const char* params_filename, 
    const int rank, double* energy_deposition_tally)
{
  double* h_energy_deposition_tally;
  allocate_host_data(&h_energy_deposition_tally, nx*ny);
  copy_buffer(nx*ny, &energy_deposition_tally, &h_energy_deposition_tally, RECV);

  double local_energy_tally = 0.0;
  for(int ii = 0; ii < nx*ny; ++ii) {
    local_energy_tally += h_energy_deposition_tally[ii];
  }

  double global_energy_tally = reduce_all_sum(local_energy_tally);

  if(rank != MASTER) {
    return;
  }

  printf("\nFinal global_energy_tally %.15e\n", global_energy_tally);

  int nresults = 0;
  char* keys = (char*)malloc(sizeof(char)*MAX_KEYS*(MAX_STR_LEN+1));
  double* values = (double*)malloc(sizeof(double)*MAX_KEYS);
  if(!get_key_value_parameter(
        params_filename, NEUTRAL_TESTS, keys, values, &nresults)) {
    printf("Warning. Test entry was not found, could NOT validate.\n");
    return;
  }

  printf("Expected %.12e, result was %.12e.\n", values[0], global_energy_tally);
  if(within_tolerance(values[0], global_energy_tally, VALIDATE_TOLERANCE)) {
    printf("PASSED validation.\n");
  }
  else {
    printf("FAILED validation.\n");
  }

  free(keys);
  free(values);
}

